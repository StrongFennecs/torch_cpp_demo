#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h> 

__global__ void add_kernel(const float* a, const float* b, float* c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

torch::Tensor add(const torch::Tensor& a, const torch::Tensor& b) {
    // some torch provided checks
    TORCH_CHECK(a.is_cuda(), "tensor a must be a cuda tensor");
    TORCH_CHECK(b.is_cuda(), "tensor b must be a cuda tensor");
    TORCH_CHECK(a.is_contiguous() && b.is_contiguous(), "tensors must be contiguous");
    TORCH_CHECK(a.sizes() == b.sizes(), "input tensors must have the same size");

    // make sure to create output on the same device
    // lots of torch functions can be accessed via torch::
    torch::Tensor res = torch::empty_like(a);  

    // use the current stream for torch
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    int n = a.numel();
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    add_kernel<<<blocks, threads, 0, stream>>>(a.data_ptr<float>(), b.data_ptr<float>(), res.data_ptr<float>(), n);

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "cuda kernel failed: ", hipGetErrorString(err));

    return res;
}