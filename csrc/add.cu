#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void add_kernel(float* a, float* b, float* c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void add(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
    int n = a.size(0);
    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), n);
}